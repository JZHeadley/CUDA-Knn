#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : KNN.cu
 Author      : jzheadley
 Version     :
 Copyright   :
 Description : CUDA compute reciprocals
 ============================================================================
 */

#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <float.h>
#include <math.h>
#include <iostream>
#include <limits.h>

#include "libarff/arff_parser.h"
#include "libarff/arff_data.h"

#include "knn.h"
using namespace std;
#define DEBUG true
#define K 3

#define MIN(a,b) (((a)<(b))?(a):(b))
#define NUM_STREAMS 4

__global__ void computeDistances(int numInstances, int numAttributes, float* dataset, float* distances)
{
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	int row = tid / numInstances; // instance1Index
//	int column = tid - ((tid / numInstances) * numInstances); //instance2Index
	int column = tid % numInstances;
	if ((tid < numInstances * numInstances))
	{
		float sum = 0;
		int instance1 = row * numAttributes;
		int instance2 = column * numAttributes;
		for (int atIdx = 0; atIdx < numAttributes - 1; atIdx++) // numAttributes -1 since we don't want to compare class in the distance because that doesn't make sense
		{
			sum += ((dataset[instance1 + atIdx] - dataset[instance2 + atIdx]) * (dataset[instance1 + atIdx] - dataset[instance2 + atIdx]));
		}
		distances[row * numInstances + column] = (float) sqrt(sum);
		distances[column * numInstances + row] = distances[row * numInstances + column]; //set the distance for the other half of the pair we just computed
	}
}

__inline__ __device__ void reduceToK(float* distancesTo, int* indexes, int k, int curSize)
{
	// we're just going to do a simple bubble sort and pretend the elements past k don't exist
	float tmp;
	unsigned char idx;
	for (int i = 0; i < curSize - 1; i++)
	{
		for (int j = 0; j < curSize - i - 1; j++)
		{
			if (distancesTo[j] > distancesTo[j + 1])
			{
				tmp = distancesTo[j];
				idx = indexes[j];
				distancesTo[j] = distancesTo[j + 1];
				indexes[j] = indexes[j + 1];
				distancesTo[j + 1] = tmp;
				indexes[j + 1] = idx;
			}
		}
	}
}
__inline__ __device__ void vote(float* distancesTo, int* predictions, int *indexes, float* dataset, int k, int numAttributes)
{
	int classVotes[32];
	for (int i = 0; i < k; i++)
	{

		int classNum = dataset[indexes[i] * numAttributes + numAttributes - 1];
		classVotes[classNum] += 1;
		if (blockIdx.x == 1 && threadIdx.x == 1)
		{
			printf("instance %i votes for the class to be %i\n", indexes[i], classNum);
		}
	}
	int finalClass;
	int mostVotes = 0;
	for (int i = 0; i < 32; i++)
	{
		if (classVotes[i] > mostVotes)
		{
			finalClass = i;
			mostVotes = classVotes[i];
		}
	}
//	for (int i = 0; i < 32; i++)
//	{
//		if (classVotes[i] == mostVotes && i != finalClass)
//		{
//			vote(distancesTo, predictions, indexes, dataset, k-1, numAttributes);
//		}
//	}
	predictions[blockIdx.x] = finalClass;
}

__global__ void knn(int* predictions, float*distances, float*dataset, int numAttributes)
{
	__shared__ int indexes[256];
	__shared__ float distancesTo[256];

	// gridDim.x is numInstances
	int bestInstanceId;
	float bestDistance = INT_MAX;
	int instanceFrom = blockIdx.x * gridDim.x;
	int distancePos;
	int rowBoundary = instanceFrom + gridDim.x - 1;
	if (blockDim.x < gridDim.x)
	{ //If we have more elements than threads we need to do an inital reduction to fit into our shared mem
		if (threadIdx.x < blockDim.x) // only want 256 threads to come into this otherwise we will go out of bounds of our shared mem
		{
			for (int i = threadIdx.x; i < gridDim.x; i += blockDim.x) // will try to make this more coalesced later
			{
				if (i == blockIdx.x) // don't need to include the diagonal
					continue;

				distancePos = instanceFrom + i;
				if (distancePos > rowBoundary)
				{ // should take care of the final elements
					break;
				}
				if (distances[distancePos] < bestDistance)
				{
					if (bestDistance != INT_MAX && blockIdx.x == 1)
					{
						printf("We have a new best distance of %f at pos %i which beats %f at pos %i\n", distances[distancePos], i, bestDistance,
								bestInstanceId);
					}
					if (blockIdx.x == 1 && bestDistance != INT_MAX)
						printf("best instanceId is %i\n", bestInstanceId);
					bestDistance = distances[distancePos];
					bestInstanceId = i;
					if (blockIdx.x == 1 && bestDistance != INT_MAX)
						printf("new best instanceId is %i\n", bestInstanceId);
				}
			}
			if (blockIdx.x == 1 && threadIdx.x != bestInstanceId)
				printf("Thread %i has best distance with instance %i\n", threadIdx.x, bestInstanceId);
			indexes[threadIdx.x] = bestInstanceId;
			if (blockIdx.x == 1 && threadIdx.x != bestInstanceId)
				printf("thread %i has best distance with instance %i\n", threadIdx.x, indexes[threadIdx.x]);

			distancesTo[threadIdx.x] = bestDistance;
		}
		__syncthreads();

		if (DEBUG && blockIdx.x == 1 && threadIdx.x == 1)
		{
			for (int i = 0; i < blockDim.x; i++)
			{
				printf("(%i, %.2f) ", indexes[i], distancesTo[i]);
			}
			printf("\n");
		}
		if (threadIdx.x < blockDim.x / 2) // only need the first half(128) of the threads to work on the 256 length shared mem arrays
		{
			int s;
			// this for should probably have the conditional of (s>>1) > k but if I do that I don't reduce enough sooo...
			// we're going with this until I find that error and just upping s back up after this for
			for (s = blockDim.x / 2; (s) > K; s >>= 1)
			{
//				if (threadIdx.x == 0 && blockIdx.x == 0)
//					printf("s is %i\n", s);
				if (threadIdx.x < s)
				{

					if (distancesTo[threadIdx.x + s] < distancesTo[threadIdx.x])
					{
//						if (DEBUG && blockIdx.x == 1)
//							printf("sharedMem[%i] with value %f WAS LESS THAN sharedMem[%i] with value %f\n", threadIdx.x + s,
//									distancesTo[threadIdx.x + s], threadIdx.x, distancesTo[threadIdx.x]);
						distancesTo[threadIdx.x] = distancesTo[threadIdx.x + s];
						indexes[threadIdx.x] = indexes[threadIdx.x + s];
						if (DEBUG)
						{
							distancesTo[threadIdx.x + s] = 0;
							indexes[threadIdx.x + s] = 0;
						}
					}
					else
					{
//						if (DEBUG && blockIdx.x == 1)
//							printf("sharedMem[%i] with value %f was not less than sharedMem[%i] with value %f\n", threadIdx.x + s,
//									distancesTo[threadIdx.x + s], threadIdx.x, distancesTo[threadIdx.x]);
						if (DEBUG)
						{
							distancesTo[threadIdx.x + s] = 0;
							indexes[threadIdx.x + s] = 0;
						}
					}
					__syncthreads();
				}
			}

			if (DEBUG && blockIdx.x == 1 && threadIdx.x == 1)
			{
				for (int i = 0; i < blockDim.x; i++)
				{
					printf("(%i, %.2f) ", indexes[i], distancesTo[i]);
				}
				printf("\n");
			}
			s *= 2;
			__syncthreads();
			if (s > K && threadIdx.x == 1)
			{ // we need to reduce it just a little more
			  // remember to change both the indexes and distancesTo arrays
//				printf("need to reduce from %i to %i\n", s, K);
				reduceToK(distancesTo, indexes, K, s);
			}
			__syncthreads();
			if (threadIdx.x == 1)
				vote(distancesTo, predictions, indexes, dataset, K, numAttributes);
		}
	}
}

int main(int argc, char* argv[])
{
	if (argc != 2)
	{
		if (argc != 3)
		{
			cout << "Usage: ./main datasets/datasetFile.arff" << endl;
			exit(0);
		}
	}

	ArffParser parser(argv[1]);

	ArffData* dataset = parser.parse();

	hipStream_t *streams = (hipStream_t*) malloc(NUM_STREAMS * sizeof(hipStream_t));
	for (int i = 0; i < NUM_STREAMS; i++) // multiple streams
		hipStreamCreate(&streams[i]);

	int numInstances = dataset->num_instances();
	int numAttributes = dataset->num_attributes();
	int* h_predictions = (int *) calloc(numInstances, sizeof(int));
	printf("We're classifying %i instances with %i attributes\n", numInstances, numAttributes);
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	float milliseconds = 0;
	int numTriangularSpaces = (numInstances * numInstances); //(numInstances * (numInstances - 1)) / 2; // don't actually need the diagonal since its all 0's so we can have numInstances-1 instead of + 1 but math is hard
	float* h_dataset, *h_distances;
	hipHostMalloc(&h_dataset, sizeof(float) * numInstances * numAttributes);
	hipHostMalloc(&h_distances, sizeof(float) * numTriangularSpaces);
	printf("numTriangularSpaces is %i\n", numTriangularSpaces);

	for (int instanceNum = 0; instanceNum < numInstances; instanceNum++)
	{
		// each 'row' will be an instances
		// each 'column' a specific attribute
		ArffInstance* instance = dataset->get_instance(instanceNum);
		for (int attributeNum = 0; attributeNum < numAttributes; attributeNum++)
		{
			h_dataset[instanceNum * numAttributes + attributeNum] = (float) instance->get(attributeNum)->operator int32();
		}

	}

	float* d_dataset;
	float* d_distances;
	int* d_predictions;

	hipMalloc(&d_predictions, numInstances * sizeof(int));
	hipMalloc(&d_dataset, numInstances * numAttributes * sizeof(float));
	hipHostMalloc(&d_distances, numTriangularSpaces * sizeof(float));

	int threadsPerBlock = 256;
//	int blocksPerGrid = (numInstances + threadsPerBlock - 1) / threadsPerBlock;
	int blocksPerGrid = ((numInstances * numInstances) + threadsPerBlock - 1) / threadsPerBlock;
	hipEventRecord(start);

	hipMemcpyAsync(d_dataset, h_dataset, numInstances * numAttributes * sizeof(float), hipMemcpyHostToDevice, streams[0]);
	hipMemcpyAsync(d_distances, h_distances, numTriangularSpaces * sizeof(float), hipMemcpyHostToDevice, streams[0]);
	computeDistances<<<blocksPerGrid, threadsPerBlock, 0, streams[0]>>>(numInstances, numAttributes, d_dataset, d_distances);
	if (DEBUG)
	{
		if (numInstances < 32)
		{
			hipMemcpyAsync(h_distances, d_distances, numTriangularSpaces * sizeof(float), hipMemcpyDeviceToHost, streams[0]);

			for (int i = 0; i < numInstances; i++)
			{
				for (int j = 0; j < numInstances; j++)
				{
					int position = (i * numInstances + j);
					printf("%.2f\t", h_distances[position]);
				}
				printf("\n");
			}
		}
	}

	hipMemcpyAsync(d_predictions, h_predictions, numInstances * sizeof(int), hipMemcpyHostToDevice, streams[1]);
	hipStreamSynchronize(streams[0]); // need this to ensure that the previous kernel computing the distances is finished otherwise we might not have the full distance matrix
	knn<<<numInstances, 256, 0, streams[1]>>>(d_predictions, d_distances, d_dataset, numAttributes);
	hipMemcpyAsync(h_predictions, d_predictions, numInstances * sizeof(int), hipMemcpyDeviceToHost, streams[1]);

	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);

	hipError_t hipError_t = hipGetLastError();

	if (hipError_t != hipSuccess)
	{
		fprintf(stderr, "hipGetLastError() returned %d: %s\n", hipError_t, hipGetErrorString(hipError_t));
		exit(EXIT_FAILURE);
	}

	int* confusionMatrix = computeConfusionMatrix(h_predictions, dataset);
	float accuracy = computeAccuracy(confusionMatrix, dataset);

	printf("The KNN classifier for %lu instances required %llu ms CPU time. Accuracy was %.4f\n", numInstances, (long long unsigned int) milliseconds,
			accuracy);

	return 0;
}

int* computeConfusionMatrix(int* predictions, ArffData* dataset)
{
	int* confusionMatrix = (int*) calloc(dataset->num_classes() * dataset->num_classes(), sizeof(int)); // matriz size numberClasses x numberClasses

	for (int i = 0; i < dataset->num_instances(); i++) // for each instance compare the true class and predicted class
	{
		int trueClass = dataset->get_instance(i)->get(dataset->num_attributes() - 1)->operator int32();
		int predictedClass = predictions[i];

		confusionMatrix[trueClass * dataset->num_classes() + predictedClass]++;
	}

	return confusionMatrix;
}

float computeAccuracy(int* confusionMatrix, ArffData* dataset)
{
	int successfulPredictions = 0;

	for (int i = 0; i < dataset->num_classes(); i++)
	{
		successfulPredictions += confusionMatrix[i * dataset->num_classes() + i]; // elements in the diagnoal are correct predictions
	}

	return successfulPredictions / (float) dataset->num_instances();
}

double euclideanDistance(ArffInstance* instance1, ArffInstance* instance2, int numAttributes)
{
	double sum = 0;
	for (int attributeIndex = 0; attributeIndex < (numAttributes - 1); attributeIndex++)
	{
		sum += pow((instance2->get(attributeIndex)->operator int32()) - (instance1->get(attributeIndex)->operator int32()), 2);
	}
	return sqrt(sum);
}

